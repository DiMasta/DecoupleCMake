
#include <hip/hip_runtime.h>
enum TextureInstruction {
	TI_INVALID = -1,
	TI_DIRT = 0,
	TI_CHECKER,
	TI_RAMP,
};

//*****************************************************************************************************************
//*****************************************************************************************************************

// COMPILE_?? flags are passed to nvrtc based on which textures must be used
__device__ int processInstruction(int instruction) {
	int textureResult = -1;

	switch (instruction) {
#ifdef COMPILE_DIRT
		extern __device__ int evalDirt();
		case TI_DIRT: {
			textureResult = evalDirt();
			break;
		}
#endif // COMPILE_DIRT

#ifdef COMPILE_CHECKER
		extern __device__ int evalChecker();
		case TI_CHECKER: {
			textureResult = evalChecker();
			break;
		}
#endif // COMPILE_CHECKER

#ifdef COMPILE_RAMP
		extern __device__ int evalRamp();
		case TI_RAMP: {
			textureResult = evalRamp();
			break;
		}
#endif // COMPILE_RAMP

		default: {
			printf("NO TEXTURE USED\n");
		}
	}

	return textureResult;
}
