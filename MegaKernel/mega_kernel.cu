
#include <hip/hip_runtime.h>
extern __device__ int processInstruction(int instruction);

extern "C" __global__ void megaKernel(int* texInstrs, size_t texInstrsCount, int* result) {
	int shadingResult = 0;
	
	for (size_t texInstrIdx = 0; texInstrIdx < texInstrsCount; ++texInstrIdx) {
		shadingResult += processInstruction(texInstrs[texInstrIdx]);
	}

	*result = shadingResult;
}